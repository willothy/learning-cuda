#include <hip/hip_runtime.h>
#include <iostream>

// Sorta following introductory series by Creel
// (https://youtube.com/@WhatsACreel)

__global__ void AddIntsCUDA(int *a, int *b) {
  // Dumb kernel that wastes a GPU thread!
  a[0] += b[0];
}

// Nice way to use cudaMalloc when experimenting with CUDA
//
// Idk if stuff like this is considered good or bad practice.
template <typename T> T *tryCudaMalloc() {
  T *ptr;
  if (hipMalloc(&ptr, sizeof(T)) != hipSuccess) {
    std::cerr << "hipMalloc failed" << std::endl;
    return nullptr;
  }
  return ptr;
}

int main() {
  int a = 5, b = 9;

  int *d_a = tryCudaMalloc<int>();
  int *d_b = tryCudaMalloc<int>();

  if (d_a == nullptr || d_b == nullptr) {
    return 1;
  }

  hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

  std::cout << "a old = " << a << std::endl;

  AddIntsCUDA<<<1, 1>>>(d_a, d_b);

  hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

  std::cout << "a new = " << a << std::endl;

  hipFree(d_a);
  hipFree(d_b);

  std::cout << "CUDA version: " << CUDA_VERSION << std::endl;
  return 0;
}
