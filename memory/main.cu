#include <ctime>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>

// Nice way to use cudaMalloc when experimenting with CUDA
//
// Idk if stuff like this is considered good or bad practice.
template <typename T> T *tryCudaMalloc(size_t count) {
  T *ptr;
  if (hipMalloc(&ptr, sizeof(T) * count) != hipSuccess) {
    std::cerr << "hipMalloc failed" << std::endl;
    return nullptr;
  }
  return ptr;
}

int main() {
  //

  return 0;
}
