#include <ctime>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>

// Nice way to use cudaMalloc when experimenting with CUDA
//
// Idk if stuff like this is considered good or bad practice.
template <typename T> T *tryCudaMalloc(size_t count) {
  T *ptr;
  if (hipMalloc(&ptr, sizeof(T) * count) != hipSuccess) {
    std::cerr << "hipMalloc failed" << std::endl;
    return nullptr;
  }
  return ptr;
}

__global__ void AddInts(int *a, int *b, int count) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  if (id < count) {
    a[id] += b[id];
  }
}

int main() {
  srand(time(NULL));

  int count = 1000;
  int *h_a = new int[count];
  int *h_b = new int[count];

  for (int i = 0; i < count; i++) {
    h_a[i] = rand() % 1000;
    h_b[i] = rand() % 1000;
  }

  std::cout << "Before" << std::endl;
  for (int i = 0; i < 5; i++) {
    std::cout << h_a[i] << " " << h_b[i] << std::endl;
  }

  int *d_a = tryCudaMalloc<int>(count);
  int *d_b = tryCudaMalloc<int>(count);
  if (d_a == nullptr || d_b == nullptr) {
    if (d_a != nullptr) {
      hipFree(d_a);
    }
    if (d_b != nullptr) {
      hipFree(d_b);
    }
    return 1;
  }

#define FREE_ALL()                                                             \
  do {                                                                         \
    hipFree(d_a);                                                             \
    hipFree(d_b);                                                             \
    delete h_a;                                                                \
    delete h_b;                                                                \
  } while (0)

  if (hipMemcpy(d_a, h_a, sizeof(int) * count, hipMemcpyHostToDevice) !=
      hipSuccess) {
    std::cout << "Memcpy failed" << std::endl;
    FREE_ALL();
    return 1;
  }
  if (hipMemcpy(d_b, h_b, sizeof(int) * count, hipMemcpyHostToDevice) !=
      hipSuccess) {
    std::cout << "Memcpy failed" << std::endl;
    FREE_ALL();
    return 1;
  }

  AddInts<<<count / 256 + 1, 256>>>(d_a, d_b, count);

  if (hipMemcpy(h_a, d_a, sizeof(int) * count, hipMemcpyDeviceToHost) !=
      hipSuccess) {
    std::cout << "Memcpy failed" << std::endl;
    FREE_ALL();
    return 1;
  }

  std::cout << "After" << std::endl;
  for (int i = 0; i < 5; i++) {
    std::cout << h_a[i] << " " << h_b[i] << std::endl;
  }

  FREE_ALL();

  return 0;
}
